#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void HelloWorld(){
    printf("hello world, %d, %d\n", blockIdx.x, threadIdx.x);
}

int main(){
    HelloWorld<<<100, 100>>>();
    hipDeviceSynchronize();
    return 0;
}